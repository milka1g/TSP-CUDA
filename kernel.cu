#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <math.h>
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include ""
#include <float.h>
#include <time.h>
#include <fstream>
#include <string>
#include <chrono>
#include <vector>
#include <sstream>


using namespace std;

#define BLOCKSIZE 256
#define PERM_SIZE 12
#define REAL_PERM 13 //max size we will need for cuda
#define THREAD_BLOCKS 1871100 //12! / 256 = 479001600/256 = 1871100

__constant__ float distanceMap[REAL_PERM + 2][REAL_PERM + 2];

int next_permutation(const int N, int* P) {
	int s;
	int* first = &P[0];
	int* last = &P[N - 1];
	int* k = last - 1;
	int* l = last;
	//find larges k so that P[k]<P[k+1]
	while (k > first) {
		if (*k < *(k + 1)) {
			break;
		}
		k--;
	}
	//if no P[k]<P[k+1], P is the last permutation in lexicographic order
	if (*k > *(k + 1)) {
		return 0;
	}
	//find largest l so that P[k]<P[l]
	while (l > k) {
		if (*l > *k) {
			break;
		}
		l--;
	}
	//swap P[l] and P[k]
	s = *k;
	*k = *l;
	*l = s;
	//reverse the remaining P[k+1]...P[N-1]
	first = k + 1;
	while (first < last) {
		s = *first;
		*first = *last;
		*last = s;

		first++;
		last--;
	}

	return 1;
}


unsigned long long factorial(int n)
{
	unsigned long long factorial = 1;
	for (int i = 1; i <= n; ++i)
	{
		factorial *= i;
	}
	return factorial;
}


int* permCPU(unsigned long long m)
{
	int i, ind;
	int* permuted = new int[REAL_PERM];
	int* elems = new int[REAL_PERM];

	for (i = 0; i < REAL_PERM; i++) elems[i] = i + 1;  //first and last hole is fixed, we permute the numHoles-2 in between

	for (i = 0; i < REAL_PERM; i++)
	{
		ind = m % (REAL_PERM - i);
		m = m / (REAL_PERM - i);
		permuted[i] = elems[ind];
		elems[ind] = elems[REAL_PERM - i - 1];
	}
	delete[] elems;
	return permuted;
}


__global__ void kernelReduce(float* distance, unsigned long long* step, unsigned int* index) {
	//extern __shared__ float shared[];
	__shared__ float distances[BLOCKSIZE];
	__shared__ unsigned int realindex[BLOCKSIZE];
	unsigned int tid = threadIdx.x;
	unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < 479001600) {
		unsigned int i, ind;
		unsigned long long m = id + (*step);
		unsigned int permuted[REAL_PERM];
		unsigned int elems[REAL_PERM];
		float len = 0;

		for (i = 0; i < REAL_PERM; i++) elems[i] = i + 1; //first and last hole is fixed, we permute the numHoles-2 in between

		for (i = 0; i < REAL_PERM; i++)
		{
			ind = m % (REAL_PERM - i);
			m = m / (REAL_PERM - i);
			permuted[i] = elems[ind];
			elems[ind] = elems[REAL_PERM - i - 1];
		}

		len = len + distanceMap[0][permuted[0]];
		for (i = 0; i < REAL_PERM - 1; i++)
			len = len + distanceMap[permuted[i]][permuted[i + 1]];
		len = len + distanceMap[permuted[REAL_PERM - 1]][REAL_PERM + 1];

		distances[tid] = len;
		realindex[tid] = id;

		__syncthreads();
		for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
			if (tid < s) {
				if (distances[tid] > distances[tid + s]) {
					distances[tid] = distances[tid + s];
					realindex[tid] = realindex[tid + s];
				}
			}
			__syncthreads();
		}

		if (tid == 0) {
			distance[blockIdx.x] = distances[0];
			index[blockIdx.x] = realindex[0];
		};
	}

}

int main(int argc, char* argv[]) {

	vector<vector<float>> holes;

	string fileName = argv[1];
	int numHoles = 0;
	int num_holes = 0; //from file first line check if correct
	try {
		std::ifstream file("C:\\Users\\mn170387d\\Desktop\\clusters\\" + fileName);
		std::string str;
		std::getline(file, str);
		std::istringstream in(str);
		in >> num_holes;
		while (std::getline(file, str)) {
			std::istringstream in(str);
			float x, y;
			in >> x >> y;
			vector<float> hole{ x, y };
			std::cout << hole[0] << ", " << hole[1] << "\n";
			holes.push_back(hole);
			numHoles++;
		}
	}
	catch (const std::exception&) {
		cout << "File doesn't exist!";
	}

	if (num_holes == numHoles) //we read from second to second last
		std::cout << "Numbers of holes are good: " << num_holes << '\n';

	//compute distances
	vector<vector<float>> distances(numHoles, vector<float>(numHoles));
	vector<int> bestPerm(numHoles);

	for (int i = 0; i < numHoles; i++) {
		for (int j = 0; j < numHoles; j++) {
			if (i == j)
				distances[i][j] = 0.0f;
			else
				distances[i][j] = sqrt(pow(holes[i][0] - holes[j][0], 2.0f) + pow(holes[i][1] - holes[j][1], 2.0f));
		}
	}

	//cout << "BR RUPA: " << numHoles << endl;

	if (numHoles <= 14) { //no need to do CUDA
		auto start = std::chrono::high_resolution_clock::now();
		int permSize = numHoles - 2; //0 1..11 12 for 13==numHoles
		float shortestPathLength = FLT_MAX;
		float currCost = 0.0f;
		int* P = new int[permSize];
		for (int i = 0; i < permSize; i++) {
			P[i] = i + 1; 
		}
	
		do {
			currCost = 0.0f;
			currCost += distances[0][P[0]];
			for (int i = 0; i < permSize - 1; i++) {
				currCost += distances[P[i]][P[i + 1]];
			}
			currCost += distances[P[permSize - 1]][numHoles - 1];
			if (currCost < shortestPathLength) {
				shortestPathLength = currCost;
				for (int i = 0; i < permSize ; i++) {
					bestPerm[i] = P[i];
				}
			}
		} while (next_permutation(permSize, P));

		cout << "Best cost: " << fixed << shortestPathLength << " mm.\n" << endl;

		ofstream out;
		out.open("C:\\Users\\mn170387d\\Desktop\\clusters\\solved" + fileName);
		out << numHoles << '\n';
		out << holes[0][0] << ' ' << holes[0][1] << " 0" << '\n';
		for (int i = 0; i < permSize; i++) {
			out << holes[i + 1][0] << ' ' << holes[i + 1][1] << ' ' << bestPerm[i] << '\n';
		}
		out << holes[numHoles - 1][0] << ' ' << holes[numHoles - 1][1] << ' ' << numHoles - 1;
		out.close();

		auto end = std::chrono::high_resolution_clock::now();
		std::chrono::duration<double, std::milli> duration = end - start;
		cout << "Time elapsed: " << duration.count() << "ms\n";
	}
	else { //doing cuda for 12+
		auto start = std::chrono::high_resolution_clock::now();
		//must copy contiguous arr to device 
		float distancesCont[REAL_PERM + 2][REAL_PERM + 2]; //first and last hole are fixed
		for (int i = 0; i < numHoles; i++) {
			for (int j = 0; j < numHoles; j++) {
				distancesCont[i][j] = distances[i][j];
			}
		}

		hipError_t err;
		// 
		err = hipMemcpyToSymbol(HIP_SYMBOL(distanceMap), distancesCont, (REAL_PERM + 2) * (REAL_PERM + 2) * sizeof(float), 0, hipMemcpyHostToDevice);
		if (err != hipSuccess) {
			std::cout << "Copying distanceMap failed\n";
		}

		float* h_distance, * d_distance;
		unsigned int* h_index, * d_index;
		unsigned long long* h_step = new unsigned long long, * d_step;
		*h_step = 0;
		err = hipMalloc(&d_step, sizeof(unsigned long long));
		if (err != hipSuccess) {
			std::cout << "hipMalloc failed d_step \n";
		}
		err = hipMemcpy(d_step, h_step, sizeof(unsigned long long), hipMemcpyHostToDevice);
		if (err != hipSuccess) {
			std::cout << "hipMemcpy failed d_step\n";
		}

		err = hipMalloc(&d_index, sizeof(unsigned int) * THREAD_BLOCKS);
		if (err != hipSuccess) {
			std::cout << "hipMalloc failed d_index\n";
		}

		h_distance = new float[THREAD_BLOCKS];
		h_index = new unsigned int[THREAD_BLOCKS];

		err = hipMalloc(&d_distance, sizeof(float) * THREAD_BLOCKS);
		if (err != hipSuccess) {
			std::cout << "hipMalloc failed d_distance\n";
		}


		float min = FLT_MAX;
		unsigned long long bestind = 0;

		for (int i = 0; i < factorial(REAL_PERM) / factorial(PERM_SIZE); i++) {
			err = hipMemcpy(d_step, h_step, sizeof(unsigned long long), hipMemcpyHostToDevice);
			if (err != hipSuccess) {
				std::cout << "hipMemcpy failed setting of step in loop\n";
			}

			kernelReduce << <THREAD_BLOCKS, BLOCKSIZE >> > (d_distance, d_step, d_index);

			hipDeviceSynchronize();
			err = hipMemcpy(h_distance, d_distance, sizeof(float) * THREAD_BLOCKS, hipMemcpyDeviceToHost);
			if (err != hipSuccess) {
				std::cout << "hipMemcpy failed in loop d_distance->h_distance\n";
			}
			err = hipMemcpy(h_index, d_index, sizeof(unsigned int) * THREAD_BLOCKS, hipMemcpyDeviceToHost);
			if (err != hipSuccess) {
				std::cout << "hipMemcpy failed in loop d_index->h_index\n";
			}
			for (int i = 0; i < THREAD_BLOCKS; i++) {
				if (h_distance[i] < min) {
					min = h_distance[i];
					bestind = h_index[i] + *h_step;
				}
			}
			(*h_step) += factorial(PERM_SIZE);
		}
		float newmin = 0;
		int* rez = permCPU(bestind);

		newmin += distancesCont[0][rez[0]];
		for (int i = 0; i < REAL_PERM - 1; i++)
			newmin = newmin + distancesCont[rez[i]][rez[i + 1]];
		newmin += distancesCont[rez[REAL_PERM - 1]][REAL_PERM + 1];

		std::cout << "Best path is:\n";
		std::cout << 0 << " ";
		for (int i = 0; i < REAL_PERM; i++) {
			std::cout << rez[i] + 1 << " ";
		}
		std::cout << REAL_PERM + 1 << " ";
		std::cout << std::endl;


		auto end = std::chrono::high_resolution_clock::now();
		std::chrono::duration<double, std::milli> duration = end - start;
		cout << "Time elapsed: " << duration.count() << "ms\n";

		std::cout << " Minimal path from kernel " << min << ", calculated " << newmin << ", execution time: " << duration.count() << "ms" << std::endl;

		ofstream out;
		out.open("C:\\Users\\mn170387d\\Desktop\\clusters\\solved" + fileName);
		out << numHoles << '\n';
		out << holes[0][0] << ' ' << holes[0][1] << " 0" << '\n';
		for (int i = 0; i < REAL_PERM; i++) {
			out << holes[i + 1][0] << ' ' << holes[i + 1][1] << ' ' << rez[i] << '\n';
		}
		out << holes[numHoles - 1][0] << ' ' << holes[numHoles - 1][1] << ' ' << numHoles - 1 << '\n';
		out.close();

		hipFree(d_distance);
		hipFree(d_index);
		hipFree(d_step);
		delete[] h_distance;
		delete h_step;
		delete[] h_index;
		delete[] rez;

		hipDeviceReset();
	}

	return 0;
}